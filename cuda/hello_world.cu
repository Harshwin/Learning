    
#include <hip/hip_runtime.h>
#include <stdio.h>

    #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
    #error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher
    #endif

    __global__ void device_greetings()
    {
      printf("Fuck you\n");
    }

    int main(void)
    {
      // greet from the host
      printf("Hello, world from the host!\n");

      // launch a kernel with a single thread to greet from the device
      device_greetings<<<1,10>>>();

      hipDeviceSynchronize();

      return 0;
    }
